#include "hip/hip_runtime.h"
#include "cpuGraph.h"
#include "gpuGraph.h"
#include "structure.cuh"
#include "cuda_helpers.h"

#include <cmath>

gpuRelation::gpuRelation()
{
  cuchk(hipMalloc(&keys_, 2 * sizeof(vtype)));
  cuchk(hipMalloc(&num_candidates_, sizeof(numtype)));
  cuchk(hipMemset(num_candidates_, 0, sizeof(numtype)));
  candidate_vs_[0] = nullptr;
  candidate_vs_[1] = nullptr;
}

gpuRelation::~gpuRelation()
{
  if (keys_)
    cuchk(hipFree(keys_));
  if (num_candidates_)
    cuchk(hipFree(num_candidates_));
  if (candidate_vs_[0])
    cuchk(hipFree(candidate_vs_[0]));
  if (candidate_vs_[1])
    cuchk(hipFree(candidate_vs_[1]));
}

void gpuRelation::copy_from_cpu(const cpuRelation &cpu_relations, bool copy_keys)
{
  hipStream_t streams[4];
  for (int i = 0; i < 4; ++i)
    hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
  if (copy_keys)
    cuchk(hipMemcpyAsync(keys_, cpu_relations.keys_, 2 * sizeof(vtype), hipMemcpyHostToDevice, streams[0]));
  cuchk(hipMemcpyAsync(num_candidates_, cpu_relations.num_candidates_, sizeof(numtype), hipMemcpyHostToDevice, streams[1]));
  cuchk(hipMemcpyAsync(candidate_vs_[0], cpu_relations.candidate_vs_[0], cpu_relations.num_candidates_[0] * sizeof(vtype), hipMemcpyHostToDevice, streams[2]));
  cuchk(hipMemcpyAsync(candidate_vs_[1], cpu_relations.candidate_vs_[1], cpu_relations.num_candidates_[0] * sizeof(vtype), hipMemcpyHostToDevice, streams[3]));
  for (int i = 0; i < 4; ++i)
    hipStreamSynchronize(streams[i]);
  for (int i = 0; i < 4; ++i)
    hipStreamDestroy(streams[i]);
}

void gpuRelation::copy_to_cpu(cpuRelation &cpu_relations, bool copy_keys)
{
  hipStream_t streams[4];
  for (int i = 0; i < 4; ++i)
    hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
  if (copy_keys)
    cuchk(hipMemcpyAsync(cpu_relations.keys_, keys_, 2 * sizeof(vtype), hipMemcpyDeviceToHost, streams[0]));
  cuchk(hipMemcpyAsync(cpu_relations.num_candidates_, num_candidates_, sizeof(numtype), hipMemcpyDeviceToHost, streams[1]));
  cuchk(hipMemcpyAsync(cpu_relations.candidate_vs_[0], candidate_vs_[0], cpu_relations.num_candidates_[0] * sizeof(vtype), hipMemcpyDeviceToHost, streams[2]));
  cuchk(hipMemcpyAsync(cpu_relations.candidate_vs_[1], candidate_vs_[1], cpu_relations.num_candidates_[0] * sizeof(vtype), hipMemcpyDeviceToHost, streams[3]));
  for (int i = 0; i < 4; ++i)
    hipStreamSynchronize(streams[i]);
  for (int i = 0; i < 4; ++i)
    hipStreamDestroy(streams[i]);
}

cpuRelation::cpuRelation()
{
  cuchk(hipHostMalloc(&keys_, sizeof(vtype) * 2));
  cuchk(hipHostMalloc(&num_candidates_, sizeof(numtype)));
  memset(num_candidates_, 0, sizeof(uint32_t));
  candidate_vs_[0] = nullptr;
  candidate_vs_[1] = nullptr;
}

cpuRelation::~cpuRelation()
{
  if (keys_)
    cuchk(hipHostFree(keys_));
  if (num_candidates_)
    cuchk(hipHostFree(num_candidates_));
  if (candidate_vs_[0])
    cuchk(hipHostFree(candidate_vs_[0]));
  if (candidate_vs_[1])
    cuchk(hipHostFree(candidate_vs_[1]));
}

void cpuRelation::copy_from_gpu(const gpuRelation &gpu_relations, bool copy_keys)
{
  hipStream_t streams[4];
  for (int i = 0; i < 4; ++i)
    hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
  if (copy_keys)
    cuchk(hipMemcpyAsync(keys_, gpu_relations.keys_, 2 * sizeof(vtype), hipMemcpyDeviceToHost, streams[0]));
  cuchk(hipMemcpy(num_candidates_, gpu_relations.num_candidates_, sizeof(numtype), hipMemcpyDeviceToHost));

  cuchk(hipHostMalloc(&candidate_vs_[0], sizeof(vtype) * num_candidates_[0]));
  cuchk(hipHostMalloc(&candidate_vs_[1], sizeof(vtype) * num_candidates_[0]));
  cuchk(hipMemcpyAsync(candidate_vs_[0], gpu_relations.candidate_vs_[0], num_candidates_[0] * sizeof(vtype), hipMemcpyDeviceToHost, streams[2]));
  cuchk(hipMemcpyAsync(candidate_vs_[1], gpu_relations.candidate_vs_[1], num_candidates_[0] * sizeof(vtype), hipMemcpyDeviceToHost, streams[3]));
  for (int i = 0; i < 4; ++i)
    hipStreamSynchronize(streams[i]);
  for (int i = 0; i < 4; ++i)
    hipStreamDestroy(streams[i]);
}

void cpuRelation::copy_to_gpu(gpuRelation &gpu_relations, bool copy_keys)
{
  hipStream_t streams[4];
  for (int i = 0; i < 4; ++i)
    hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
  if (copy_keys)
    cuchk(hipMemcpyAsync(gpu_relations.keys_, keys_, 2 * sizeof(vtype), hipMemcpyHostToDevice, streams[0]));
  cuchk(hipMemcpyAsync(gpu_relations.num_candidates_, num_candidates_, sizeof(numtype), hipMemcpyHostToDevice, streams[1]));
  cuchk(hipMemcpyAsync(gpu_relations.candidate_vs_[0], candidate_vs_[0], num_candidates_[0] * sizeof(vtype), hipMemcpyHostToDevice, streams[2]));
  cuchk(hipMemcpyAsync(gpu_relations.candidate_vs_[1], candidate_vs_[1], num_candidates_[0] * sizeof(vtype), hipMemcpyHostToDevice, streams[3]));
  for (int i = 0; i < 4; ++i)
    hipStreamSynchronize(streams[i]);
  for (int i = 0; i < 4; ++i)
    hipStreamDestroy(streams[i]);
}

cpuCluster::cpuCluster()
{
  num_query_us = 0;
  query_us_ = nullptr;
}

cpuCluster::~cpuCluster()
{
  if (query_us_)
    delete[] query_us_;
}

cpuCluster &cpuCluster::operator=(const cpuCluster &rhs)
{
  num_query_us = rhs.num_query_us;
  query_us_ = new vtype[num_query_us];
  memcpy(query_us_, rhs.query_us_, num_query_us * sizeof(vtype));
  return *this;
}

gpuCluster::gpuCluster()
{
  num_query_us = 0;
  query_us_ = nullptr;
}

gpuCluster::~gpuCluster()
{
  if (query_us_)
    cuchk(hipFree(query_us_));
}

encodingMeta::encodingMeta()
{
  num_clusters = 0;
  num_query_us_ = nullptr;
  num_total_us = 0;
  num_blocks = 0;
  query_us_compact_ = nullptr;
  cluster_offsets_ = nullptr;
  is_a_valid_cluster_ = new bool[MAX_CLUSTERS];
  memset(is_a_valid_cluster_, true, sizeof(bool) * MAX_CLUSTERS);

  // layer
  num_layers = 0;
  num_clusters_per_layer_ = new numtype[MAX_LAYERS];
  memset(num_clusters_per_layer_, 0, sizeof(numtype) * MAX_LAYERS);
  layer_offsets_ = new offtype[MAX_LAYERS];
  memset(layer_offsets_, 0, sizeof(offtype) * MAX_LAYERS);

  // merge
  merge_count = 0;
  merged_cluster_left_ = nullptr;
  merged_cluster_right_ = nullptr;
  merged_cluster_vertex_ = nullptr;
  merged_cluster_layer_ = nullptr;

  // combine
  combine_cnt = 0;
  combine_clusters_other_ = nullptr;
  combine_cluster_out_ = nullptr;
  combine_type_ = nullptr;

  combine_checkpoints_ = new int[MAX_LAYERS];
  memset(combine_checkpoints_, -1, sizeof(int) * MAX_LAYERS);
}

encodingMeta::~encodingMeta()
{
  if (num_query_us_)
    delete[] num_query_us_;
  if (query_us_compact_)
    delete[] query_us_compact_;
  if (cluster_offsets_)
    delete[] cluster_offsets_;

  if (is_a_valid_cluster_)
    delete[] is_a_valid_cluster_;

  // layer
  if (num_clusters_per_layer_)
    delete[] num_clusters_per_layer_;

  // merge
  if (merged_cluster_left_)
    delete[] merged_cluster_left_;
  if (merged_cluster_right_)
    delete[] merged_cluster_right_;
  if (merged_cluster_vertex_)
    delete[] merged_cluster_vertex_;
  if (merged_cluster_layer_)
    delete[] merged_cluster_layer_;

  // combine
  if (combine_clusters_other_)
    delete[] combine_clusters_other_;
  if (combine_cluster_out_)
    delete[] combine_cluster_out_;
  if (combine_type_)
    delete[] combine_type_;
}

void encodingMeta::init(cpuCluster *cpu_clusters_)
{
  num_query_us_ = new numtype[num_clusters];
  cluster_offsets_ = new numtype[num_clusters + 1];
  num_total_us = 0;
  for (int i = 0; i < num_clusters; ++i)
  {
    num_query_us_[i] = cpu_clusters_[i].num_query_us;
    cluster_offsets_[i] = num_total_us;
    num_total_us += num_query_us_[i];
  }
  cluster_offsets_[num_clusters] = num_total_us;
  num_blocks = std::ceil(num_total_us / 32.0);
  query_us_compact_ = new vtype[num_total_us];
  offtype off = 0;
  for (int i = 0; i < num_clusters; ++i)
  {
    memcpy(query_us_compact_ + off, cpu_clusters_[i].query_us_, num_query_us_[i] * sizeof(vtype));
    off += num_query_us_[i];
  }
}

void encodingMeta::print()
{
  std::cout << "num_clusters: " << num_clusters << std::endl;
  std::cout << "num_total_us: " << num_total_us << std::endl;
  std::cout << "num_blocks: " << num_blocks << std::endl;
  std::cout << "num_query_us_ in each cluster: ";
  for (int i = 0; i < num_clusters; ++i)
    std::cout << num_query_us_[i] << " ";
  std::cout << std::endl;
  std::cout << "cluster_offsets: ";
  for (int i = 0; i < num_clusters + 1; ++i)
    std::cout << cluster_offsets_[i] << " ";
  std::cout << std::endl;
  std::cout << "query_us_compact_: ";
  for (int i = 0; i < num_total_us; ++i)
    std::cout << query_us_compact_[i] << " ";
  std::cout << std::endl;
}