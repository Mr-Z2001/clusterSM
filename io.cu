#include "hip/hip_runtime.h"
#include "io.cuh"
#include "globals.cuh"
#include "cuda_helpers.h"

#include <iostream>
#include <fstream>
#include <string>
#include <cstring>
#include <unordered_set>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <unordered_map>

using std::cerr;
using std::cout;
using std::endl;

// no `eid` in the graph file.
// for an undirected edge, appear only once in the file. So we should store each edge twice.
void readGraphToCPU2(
		cpuGraph *graph,
		const char *filename)
{
	std::ifstream ifs(filename);
	if (!ifs.is_open())
	{
		cerr << "Error: Unable to open file " << filename << endl;
		exit(1);
	}
	char type;
	numtype num_v, num_e;
	ifs >> type >> num_v >> num_e;
	if (type != 't')
	{
		cerr << "Error: Expected 't' at the beginning \n"
				 << "While found " << type << endl;
		exit(1);
	}

	graph->num_v = num_v;
	graph->num_e = num_e;
	graph->largest_l = 0;

	graph->vertexIDs_ = new vtype[graph->num_v];
	graph->offsets_ = new offtype[graph->num_v + 1];
	graph->neighbors_ = new vtype[graph->num_e << 1];
	graph->edgeIDs_ = new etype[graph->num_e << 1];

	graph->outdeg_ = new degtype[graph->num_v];
	graph->vLabels_ = new vltype[graph->num_v];
	memset(graph->outdeg_, 0, sizeof(degtype) * graph->num_v);

	offtype *offs = new offtype[graph->num_v];
	memset(offs, 0, sizeof(offtype) * (graph->num_v));
	std::unordered_map<vltype, numtype> labelCount;
	graph->offsets_[0] = 0;
	etype eid_global = 0;

	while (ifs >> type)
	{
		if (type == 'v')
		{
			vtype vid;
			vltype vLabel;
			degtype deg;
			ifs >> vid >> vLabel >> deg;
			graph->vertexIDs_[vid] = vid;
			graph->vLabels_[vid] = vLabel;
			graph->outdeg_[vid] = deg;
			labelCount[vLabel]++;
			graph->offsets_[vid + 1] = graph->offsets_[vid] + deg;
		}
		else
		{
			vtype src, dst;
			// etype eid = eid_global++;
			ifs >> src >> dst;
			graph->vve.insert({std::pair(src, dst), eid_global});
			graph->evv.insert(std::make_pair(eid_global, std::make_pair(src, dst)));
			graph->vve.insert({std::pair(dst, src), eid_global + 1});
			graph->evv.insert(std::make_pair(eid_global + 1, std::make_pair(dst, src)));
			offtype off = graph->offsets_[src] + offs[src];
			graph->neighbors_[off] = dst;
			graph->edgeIDs_[off] = eid_global;
			offs[src]++;
			off = graph->offsets_[dst] + offs[dst];
			graph->neighbors_[off] = src;
			graph->edgeIDs_[off] = eid_global + 1;
			offs[dst]++;
			eid_global += 2;
		}
	}
	delete[] offs;
	for (auto l_c : labelCount)
	{
		graph->maxLabelFreq = std::max(graph->maxLabelFreq, l_c.second);
		graph->largest_l = std::max(graph->largest_l, l_c.first);
	}
	for (int i = 0; i < graph->num_v; i++)
		graph->maxDegree = std::max(graph->maxDegree, graph->outdeg_[i]);

	graph->keep = new bool[graph->num_e * 2];
	memset(graph->keep, false, sizeof(bool) * graph->num_e * 2);
}

// Up to 64 edges for query graph.
// If query graph is a complete graph, up to 11 vertices.

// `eid` in the file.
// for undirected edges appear twice in the file.
void readGraphToCPU(
		cpuGraph *graph,
		const char *filename)
{
	std::ifstream
			ifs(filename);
	if (!ifs.is_open())
	{
		cerr
				<< "Error: Unable to open file "
				<< filename
				<< endl;
		exit(1);
	}

	char type;
	numtype num_v, num_e;
	ifs >> type >> num_v >> num_e;
	if (type != 't')
	{
		cerr
				<< "Error: Expected 't' at the beginning \n"
				<< "While found " << type
				<< endl;
		exit(1);
	}

	// init.
	graph->num_v = num_v;
	graph->num_e = num_e;
	graph->largest_l = 0;

	graph->vertexIDs_ = new vtype[graph->num_v];
	graph->offsets_ = new offtype[graph->num_v + 1];
	graph->neighbors_ = new vtype[graph->num_e << 1];
	graph->edgeIDs_ = new etype[graph->num_e << 1];

	graph->outdeg_ = new degtype[graph->num_v];
	graph->vLabels_ = new vltype[graph->num_v];
	memset(graph->outdeg_, 0, sizeof(degtype) * graph->num_v);

	// v_id starts from 0 and is continuous
	vtype v_id;
	vltype vLabel;
	std::unordered_map<vltype, numtype> labelCount;
	for (vtype i = 0; i < graph->num_v; ++i)
	{
		ifs >> type >> v_id >> vLabel;
		if (type != 'v')
		{
			std::cerr << "Error: Invalid vertex label" << std::endl;
			exit(1);
		}
		graph->vertexIDs_[i] = v_id;
		graph->vLabels_[v_id] = vLabel;
		labelCount[vLabel]++;
	}
	for (auto l_c : labelCount)
	{
		graph->largest_l = std::max(graph->largest_l, l_c.first);
		graph->maxLabelFreq = std::max(graph->maxLabelFreq, l_c.second);
	}

	// edgeID grows from 0.

	offtype off = 0; // point to both `neighbors_` and `edgeIDs_`
	vtype lastSrc = 0, src, dst;
	etype eid;
	graph->offsets_[0] = 0;
	// cout << (graph->num_e << 1) << endl;
	for (etype i = 0; i < (graph->num_e << 1); ++i)
	{
		ifs >> type >> src >> dst >> eid;
		if (type != 'e')
		{
			std::cerr << "Error: Expected 'e' at the beginning. \n"
								<< "while found " << type
								<< std::endl;
			exit(1);
		}
		graph->vve.insert({std::pair(src, dst), eid});

		if (src == dst)
			continue;
		graph->neighbors_[off] = dst;
		graph->edgeIDs_[off] = eid;
		graph->outdeg_[src]++;
		// // graph->eLabels[off] = label;

		if (src != lastSrc)
		{
			graph->offsets_[src] = off;
			lastSrc = src;
		}

		++off;
	}
	graph->offsets_[graph->num_v] = off;
	// cout << "edge done" << endl; // pass
	for (offtype u_off = 0; u_off < graph->num_v; ++u_off)
		graph->maxDegree = std::max(graph->maxDegree, graph->outdeg_[u_off]);

	// cout << "read to cpu all done" << endl; // pass
}

void allocateMemGPU(
		gpuGraph *gpuGraph,
		const cpuGraph *cpuGraph)
{
	uint32_t num = cpuGraph->num_v;
	if (cpuGraph->isQuery)
		num = NUM_VQ;
	cuchk(hipMalloc((void **)&gpuGraph->degree_, sizeof(degtype) * num));
	cuchk(hipMalloc((void **)&gpuGraph->vLabels_, sizeof(vltype) * num));

	cuchk(hipMalloc((void **)&gpuGraph->offsets_, sizeof(offtype) * (num + 1)));
	cuchk(hipMalloc((void **)&gpuGraph->neighbors_, sizeof(vtype) * cpuGraph->num_e * 2));
	cuchk(hipMalloc((void **)&gpuGraph->edgeIDs_, sizeof(etype) * cpuGraph->num_e * 2));
}

void copyGraphToGPU(
		gpuGraph *gpuGraph,
		const cpuGraph *cpuGraph)
{
	uint32_t num = cpuGraph->num_v;
	if (cpuGraph->isQuery)
		num = NUM_VQ;
	cuchk(hipMemcpy(gpuGraph->degree_, cpuGraph->outdeg_, sizeof(uint32_t) * num, hipMemcpyHostToDevice));
	cuchk(hipMemcpy(gpuGraph->vLabels_, cpuGraph->vLabels_, sizeof(vltype) * num, hipMemcpyHostToDevice));
	// cuchk(hipMemcpy(gpuGraph->eLabels, cpuGraph->eLabels, sizeof(eltype) * cpuGraph->num_e * 2, hipMemcpyHostToDevice));

	cuchk(hipMemcpy(gpuGraph->offsets_, cpuGraph->offsets_, sizeof(uint32_t) * (num + 1), hipMemcpyHostToDevice));
	cuchk(hipMemcpy(gpuGraph->neighbors_, cpuGraph->neighbors_, sizeof(vtype) * cpuGraph->num_e * 2, hipMemcpyHostToDevice));
	cuchk(hipMemcpy(gpuGraph->edgeIDs_, cpuGraph->edgeIDs_, sizeof(etype) * cpuGraph->num_e * 2, hipMemcpyHostToDevice));
}

void copyGraphToCPU(
		gpuGraph *gpuGraph,
		cpuGraph *cpuGraph)
{
	uint32_t num = NUM_VQ;
	cuchk(hipMemcpy(cpuGraph->outdeg_, gpuGraph->degree_, sizeof(uint32_t) * num, hipMemcpyDeviceToHost));
	cuchk(hipMemcpy(cpuGraph->vLabels_, gpuGraph->vLabels_, sizeof(vltype) * num, hipMemcpyDeviceToHost));
	// cuchk(hipMemcpy(cpuGraph->eLabels, gpuGraph->eLabels, sizeof(eltype) * cpuGraph->num_e * 2, hipMemcpyDeviceToHost));

	cuchk(hipMemcpy(cpuGraph->offsets_, gpuGraph->offsets_, sizeof(uint32_t) * (num + 1), hipMemcpyDeviceToHost));
	cuchk(hipMemcpy(cpuGraph->neighbors_, gpuGraph->neighbors_, sizeof(vtype) * cpuGraph->num_e * 2, hipMemcpyDeviceToHost));
	cuchk(hipMemcpy(cpuGraph->edgeIDs_, gpuGraph->edgeIDs_, sizeof(etype) * cpuGraph->num_e * 2, hipMemcpyDeviceToHost));
}

void copyMeta(cpuGraph *query, cpuGraph *data)
{
	NUM_VQ = query->num_v;
	NUM_EQ = query->num_e;
	NUM_VLQ = query->largest_l + 1;

	// make sure that label corresponding to max_l_freq appears in the query graph.
	// labels that not in the query are invalid, useless.
	std::unordered_set<vltype> valid_vLabels;
	for (int i = 0; i < query->num_v; ++i)
		valid_vLabels.insert(query->vLabels_[i]);
	std::map<vltype, numtype> labelMap;
	for (int i = 0; i < data->num_v; ++i)
		if (valid_vLabels.find(data->vLabels_[i]) != valid_vLabels.end())
			labelMap[data->vLabels_[i]]++;
	numtype maxFreq = 0;
	for (auto l_c : labelMap)
		maxFreq = std::max(maxFreq, l_c.second);

	MAX_L_FREQ = maxFreq;
	MAX_DEGREE = data->maxDegree;

	NUM_VD = data->num_v;
	NUM_ED = data->num_e;
	// NUM_VLD = data->largest_l;
	// NUM_ELD = data->elCount;

	cuchk(hipMemcpyToSymbol(HIP_SYMBOL(C_NUM_VQ), &NUM_VQ, sizeof(uint32_t)));
	cuchk(hipMemcpyToSymbol(HIP_SYMBOL(C_NUM_EQ), &NUM_EQ, sizeof(uint32_t)));
	cuchk(hipMemcpyToSymbol(HIP_SYMBOL(C_NUM_VLQ), &NUM_VLQ, sizeof(uint32_t)));
	// cuchk(hipMemcpyToSymbol(HIP_SYMBOL(C_NUM_ELQ), &NUM_ELQ, sizeof(uint32_t)));
	cuchk(hipMemcpyToSymbol(HIP_SYMBOL(C_MAX_L_FREQ), &MAX_L_FREQ, sizeof(uint32_t)));
	cuchk(hipMemcpyToSymbol(HIP_SYMBOL(C_MAX_DEGREE), &MAX_DEGREE, sizeof(uint32_t)));

	cuchk(hipMemcpyToSymbol(HIP_SYMBOL(C_NUM_VD), &NUM_VD, sizeof(uint32_t)));
	cuchk(hipMemcpyToSymbol(HIP_SYMBOL(C_NUM_ED), &NUM_ED, sizeof(uint32_t)));
	// hipMemcpyToSymbol(HIP_SYMBOL(&C_NUM_VLD), &NUM_VLD, sizeof(uint32_t));
	// hipMemcpyToSymbol(HIP_SYMBOL(&C_NUM_ELD), &NUM_ELD, sizeof(uint32_t));
}